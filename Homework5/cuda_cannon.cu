/*
 * Damian Franco
 * CS-542
 * Homework 5
 *
 * This program implments the Copy to CPU and
 * Cuda-Aware GPU-Accelerated Cannon’s Algorithm.
 * https://jrtechs.net/data-science/cuda-vs-cpu-performance 
 */
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <time.h>
#include <math.h>

#include "mpi_cannon.hpp"

void copy_to_cpu_cannon(float* A, float* B, float* C,
        int n, int sq_num_procs, int rank_row, int rank_col)
{
    // calculate the size of the matrices in bytes
    size_t sizeA = numRowsA * numColsA * sizeof(float);
    size_t sizeB = numColsA * numColsB * sizeof(float);
    size_t sizeC = numRowsA * numColsB * sizeof(float);

    // allocate memory on the GPU for the matrices
    float* d_A, * d_B, * d_C;
    hipMalloc((void**) &d_A, sizeA);
    hipMalloc((void**) &d_B, sizeB);
    hipMalloc((void**) &d_C, sizeC);

    // copy the matrices from host memory to the GPU
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    // calculate the dimensions of the grid and blocks for the kernel
    dim3 gridDim((numColsB + BLOCK_SIZE - 1) / BLOCK_SIZE, (numRowsA + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    // launch the kernel
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, numRowsA, numColsA, numColsB);

    // copy the result matrix from the GPU to host memory
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    // free the memory allocated on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}




void cuda_aware_cannon(float* A, float* B, float* C,
        int n, int sq_num_procs, int rank_row, int rank_col)
{
    int rank, num_procs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    int size = n*n;

    float* send_A = new float[size];
    float* recv_A = new float[size];
    float* send_B = new float[size];
    float* recv_B = new float[size];

    int send_proc_A, send_proc_B;
    int recv_proc_A, recv_proc_B;
    int tag_a = 1234;
    int tag_b = 4321;

    memset(C, 0, size*sizeof(float));

    // Initial Shift : 
    get_init_procs(rank_row, rank_col, sq_num_procs,
            &send_proc_A, &send_proc_B, &recv_proc_A, &recv_proc_B);
    communicate(send_proc_A, recv_proc_A, tag_a, size, 
            rank_row && rank_col / rank_row % 2 == 0, A, recv_A);
    communicate(send_proc_B, recv_proc_B, tag_b, size, 
            rank_col && rank_row / rank_col % 2 == 0, B, recv_B);
    matmat(n, recv_A, recv_B, C);

    // Send and recv A and B from neighborhing processes in proc grid
    get_rotation_procs(rank_row, rank_col, sq_num_procs,
            &send_proc_A, &send_proc_B, &recv_proc_A, &recv_proc_B);
    for (int i = 1; i < sq_num_procs; i++)
    {
        swap(&send_A, &recv_A, &send_B, &recv_B);
        communicate(send_proc_A, recv_proc_A, tag_a, size, rank_col % 2 == 0,
                send_A, recv_A);
        communicate(send_proc_B, recv_proc_B, tag_b, size, rank_row % 2 == 0,
                send_B, recv_B);
        matmat(n, recv_A, recv_B, C);
    }

    delete[] send_A;
    delete[] recv_A;
    delete[] send_B;
    delete[] recv_B;
}